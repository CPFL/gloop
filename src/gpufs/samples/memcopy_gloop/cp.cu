#include "hip/hip_runtime.h"

#include "fs_calls.cu.h"
#include "gloop.h"
__device__ int OK;
__shared__ int zfd,zfd1, zfd2, close_ret;

template<typename Callback>
__device__ void perform_copy(uchar* scratch, size_t me, size_t filesize, const Callback& callback)
{
    if (me < filesize) {
        int toRead=min((unsigned int)FS_BLOCKSIZE,(unsigned int)(filesize-me));
        gloop::read(zfd, me, toRead, scratch, [=](size_t read) {
            if (toRead!=read) {
                assert(NULL);
            }

            gloop::write(zfd1,me,toRead,scratch, [=](size_t written) {
                if (toRead!=written) {
                    assert(NULL);
                }
                perform_copy(scratch, me + FS_BLOCKSIZE*gridDim.x, filesize, callback);
            });
        });
        return;
    }
    callback();
}

__device__ LAST_SEMAPHORE sync_sem;
__global__ void test_cpy(char* src, char* dst)
{
    __shared__ uchar* scratch;
    BEGIN_SINGLE_THREAD
        scratch=(uchar*)malloc(FS_BLOCKSIZE);
        GPU_ASSERT(scratch!=NULL);
    END_SINGLE_THREAD

    gloop::open(src,O_GRDONLY, [=](int zfd) {
        gloop::open(dst,O_GWRONCE, [=](int zfd1) {
            gloop::fstat(zfd, [=](size_t filesize) {
                size_t me=blockIdx.x*FS_BLOCKSIZE;
                perform_copy(scratch, me, filesize, [=] () {
                    gloop::close(zfd, [=](int err) {
                        gloop::close(zfd1, [=](int err) {
                        });
                    });
                });
            });
        });
    });
}

void init_device_app(){
    CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize,1<<30));
}

void init_app()
{
    void* d_OK;
    CUDA_SAFE_CALL(hipGetSymbolAddress(&d_OK,HIP_SYMBOL(OK)));
    CUDA_SAFE_CALL(hipMemset(d_OK,0,sizeof(int)));
    // INITI LOCK
    void* inited;


    CUDA_SAFE_CALL(hipGetSymbolAddress(&inited,HIP_SYMBOL(sync_sem)));
    CUDA_SAFE_CALL(hipMemset(inited,0,sizeof(LAST_SEMAPHORE)));
}

double post_app(double time, int trials){
    int res;
    CUDA_SAFE_CALL(hipMemcpyFromSymbol(&res,HIP_SYMBOL(OK),sizeof(int),0,hipMemcpyDeviceToHost));
    if(res!=0) fprintf(stderr,"Test Failed, error code: %d \n",res);
    else  fprintf(stderr,"Test Success\n");

    return 0;
}

