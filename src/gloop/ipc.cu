#include "hip/hip_runtime.h"
/*
  Copyright (C) 2016 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <cassert>
#include <gpufs/libgpufs/util.cu.h>
#include "ipc.cuh"

namespace gloop {

__host__ __device__ void IPC::emit(Code code)
{
#if defined(__CUDA_ARCH__)
    __threadfence_system();
    m_request.code = static_cast<int32_t>(code);
    __threadfence_system();
#else
    __sync_synchronize();
    m_request.code = static_cast<int32_t>(code);
    __sync_synchronize();
#endif
}

__device__ __host__ Code IPC::peek()
{
    return static_cast<Code>(m_request.code);
}

#if 0
__device__ void IPC::lock()
{
    MUTEX_LOCK(m_lock);
}

__device__ void IPC::unlock()
{
    MUTEX_UNLOCK(m_lock);
}
#endif

}  // namespace gloop
