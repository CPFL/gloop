/*
  Copyright (C) 2015 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <boost/asio.hpp>
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include "command.h"
#include "config.h"
#include "host_loop.cuh"
#include "make_unique.h"
#include "monitor_session.h"
namespace gloop {

HostLoop::HostLoop(volatile GPUGlobals* globals)
    : m_globals(globals)
    , m_loop(uv_loop_new())
    , m_socket(m_ioService)
{
    // Connect to the gloop monitor.
    {
        m_socket.connect(boost::asio::local::stream_protocol::endpoint(GLOOP_ENDPOINT));
        Command command = {
            .type = Command::Type::Initialize,
        };
        Command result { };
        while (true) {
            boost::system::error_code error;
            boost::asio::write(
                m_socket,
                boost::asio::buffer(reinterpret_cast<const char*>(&command), sizeof(Command)),
                boost::asio::transfer_all(),
                error);
            if (error != boost::asio::error::make_error_code(boost::asio::error::interrupted)) {
                break;
            }
            // retry
        }
        while (true) {
            boost::system::error_code error;
            boost::asio::read(
                m_socket,
                boost::asio::buffer(reinterpret_cast<char*>(&result), sizeof(Command)),
                boost::asio::transfer_all(),
                error);
            if (error != boost::asio::error::make_error_code(boost::asio::error::interrupted)) {
                break;
            }
        }
        m_id = result.payload;
    }
    m_requestQueue = monitor::Session::createQueue(GLOOP_SHARED_REQUEST_QUEUE, m_id, false);
    m_responseQueue = monitor::Session::createQueue(GLOOP_SHARED_RESPONSE_QUEUE, m_id, false);

    runPoller();
}

HostLoop::~HostLoop()
{
    uv_loop_close(m_loop);
    stopPoller();
}

// GPU RPC poller.
void HostLoop::runPoller()
{
    assert(!m_poller);
    m_stop.store(false, std::memory_order_release);
    m_poller = make_unique<std::thread>([this]() {
        pollerMain();
    });
}

void HostLoop::stopPoller()
{
    m_stop.store(true, std::memory_order_release);
    if (m_poller) {
        m_poller->join();
        m_poller.reset();
    }
}

void HostLoop::pollerMain()
{
    Command command = {
        .type = Command::Type::Operation,
        .payload = Command::Operation::Complete,
    };
    m_responseQueue->send(&command, sizeof(Command), 0);
    while (!m_stop.load(std::memory_order_acquire)) {
    }
}

void HostLoop::wait()
{
    while (true) {
        Command result = { };
        unsigned int priority { };
        std::size_t size { };
        m_responseQueue->receive(&result, sizeof(Command), size, priority);
        if (handle(result)) {
            break;
        }
    }
}

bool HostLoop::handle(Command command)
{
    return true;
}

}  // namespace gloop
