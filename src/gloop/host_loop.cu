/*
  Copyright (C) 2015 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <boost/asio.hpp>
#include <boost/bind.hpp>
#include <boost/thread.hpp>
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <gpufs/libgpufs/fs_initializer.cu.h>
#include <gpufs/libgpufs/host_loop.h>
#include <memory>
#include <sys/mman.h>
#include "bitwise_cast.h"
#include "command.h"
#include "config.h"
#include "data_log.h"
#include "helper.cuh"
#include "host_loop.cuh"
#include "io.cuh"
#include "ipc.cuh"
#include "make_unique.h"
#include "memcpy_io.cuh"
#include "monitor_session.h"
#include "request.h"
#include "sync_read_write.h"
#include "system_initialize.h"
#include "utility.h"
namespace gloop {

__device__ IPC* g_channel;

HostLoop::HostLoop(int deviceNumber)
    : m_deviceNumber(deviceNumber)
    , m_loop(uv_loop_new())
    , m_ioService()
    , m_monitorConnection(m_ioService)
    , m_kernelLock(*this)
{
    // Connect to the gloop monitor.
    {
        m_monitorConnection.connect(boost::asio::local::stream_protocol::endpoint(GLOOP_ENDPOINT));
        Command command = {
            .type = Command::Type::Initialize,
        };
        Command result { };
        while (true) {
            boost::system::error_code error;
            boost::asio::write(
                m_monitorConnection,
                boost::asio::buffer(reinterpret_cast<const char*>(&command), sizeof(Command)),
                boost::asio::transfer_all(),
                error);
            if (error != boost::asio::error::make_error_code(boost::asio::error::interrupted)) {
                break;
            }
            // retry
        }
        while (true) {
            boost::system::error_code error;
            boost::asio::read(
                m_monitorConnection,
                boost::asio::buffer(reinterpret_cast<char*>(&result), sizeof(Command)),
                boost::asio::transfer_all(),
                error);
            if (error != boost::asio::error::make_error_code(boost::asio::error::interrupted)) {
                break;
            }
        }
        m_id = result.payload;
    }
    m_requestQueue = monitor::Session::createQueue(GLOOP_SHARED_REQUEST_QUEUE, m_id, false);
    m_responseQueue = monitor::Session::createQueue(GLOOP_SHARED_RESPONSE_QUEUE, m_id, false);
    m_sharedMemory = monitor::Session::createMemory(GLOOP_SHARED_MEMORY, m_id, GLOOP_SHARED_MEMORY_SIZE, false);
    m_signal = make_unique<boost::interprocess::mapped_region>(*m_sharedMemory.get(), boost::interprocess::read_write, /* Offset. */ 0, GLOOP_SHARED_MEMORY_SIZE);
    GLOOP_DEBUG("id:(%u)\n", m_id);
}

HostLoop::~HostLoop()
{
    uv_loop_close(m_loop);
    stopPoller();
}

std::unique_ptr<HostLoop> HostLoop::create(int deviceNumber)
{
    gloop::initialize();
    std::unique_ptr<HostLoop> hostLoop(new HostLoop(deviceNumber));
    hostLoop->initialize();
    return hostLoop;
}

// GPU RPC poller.
void HostLoop::runPoller()
{
    assert(!m_poller);
    m_poller = make_unique<boost::thread>([this]() {
        pollerMain();
    });
}

void HostLoop::stopPoller()
{
    if (m_poller) {
        m_poller->interrupt();
        m_poller->join();
        m_poller.reset();
    }
}

void HostLoop::pollerMain()
{
    while (true) {
        if (m_currentContext) {
            if (IPC* ipc = m_currentContext->tryPeekRequest()) {
                request::Request req { };
                memcpyIO(&req, ipc->request(), sizeof(request::Request));
                ipc->emit(Code::None);
                handleIO({
                    .type = Command::Type::IO,
                    .payload = bitwise_cast<uintptr_t>(ipc),
                    .request = req,
                });
                continue;
            }
        }
        boost::this_thread::interruption_point();
    }
}

void HostLoop::initialize()
{
    {
        // This ensures that primary GPU context is initialized.
        std::lock_guard<KernelLock> lock(m_kernelLock);
        GLOOP_CUDA_SAFE_CALL(hipStreamCreate(&m_pgraph));

        GLOOP_CUDA_SAFE_CALL(hipHostRegister(m_signal->get_address(), GLOOP_SHARED_MEMORY_SIZE, hipHostRegisterMapped));
        GLOOP_CUDA_SAFE_CALL(hipHostGetDevicePointer(&m_deviceSignal, m_signal->get_address(), 0));

        for (int i = 0; i < GLOOP_THREAD_GROUP_SIZE; ++i) {
            m_copyWorkPool.release(CopyWork::create());
        }

        CUDA_SAFE_CALL(hipPeekAtLastError());
    }
}

void HostLoop::drain()
{
    // Host main loop.
#if 0
    // Run in main thread.
    m_ioService.run();
#else
    // Since kernel work is already held by kernel executing thread,
    // when joining threads, we can say that all the events produced by ASIO
    // is already drained.
    boost::thread_group threadGroup;
    for (int i = 0; i < GLOOP_THREAD_GROUP_SIZE; ++i) {
        threadGroup.create_thread(boost::bind(&boost::asio::io_service::run, &m_ioService));
    }
    threadGroup.join_all();
#endif
}

void HostLoop::prepareForLaunch()
{
    m_currentContext->prepareForLaunch();
    syncWrite<uint32_t>(static_cast<volatile uint32_t*>(m_signal->get_address()), 0);
}

void HostLoop::resume()
{
    std::lock_guard<KernelLock> lock(m_kernelLock);
    prepareForLaunch();
    tryLaunch([&] {
        gloop::resume<<<m_currentContext->blocks(), m_threads, 0, m_pgraph>>>(m_deviceSignal, m_currentContext->deviceContext());
    });
    GLOOP_CUDA_SAFE_CALL(hipStreamSynchronize(m_pgraph));
}

void HostLoop::prologue(HostContext& hostContext, dim3 threads)
{
    m_threads = threads;
    m_currentContext = &hostContext;
    runPoller();
}

void HostLoop::epilogue()
{
    stopPoller();
    logGPUfsDone();
    m_currentContext = nullptr;
}

bool HostLoop::handleIO(Command command)
{
    assert(command.type == Command::Type::IO);
    request::Request req = command.request;
    IPC* ipc = bitwise_cast<IPC*>(command.payload);

    switch (static_cast<Code>(req.code)) {
    case Code::Open: {
        int fd = m_currentContext->table().open(req.u.open.filename.data, req.u.open.mode);
        // GLOOP_DEBUG("open:(%s),fd:(%d)\n", req.u.open.filename.data, fd);
        ipc->request()->u.openResult.fd = fd;
        ipc->emit(Code::Complete);
        break;
    }

    case Code::Write: {
        // FIXME: Significant naive implementaion.
        // We should integrate implementation with GPUfs's buffer cache.
        m_ioService.post([ipc, req, this]() {
            // GLOOP_DEBUG("Write fd:(%d),count:(%u),offset:(%d),page:(%p)\n", req.u.write.fd, (unsigned)req.u.write.count, (int)req.u.write.offset, (void*)req.u.read.buffer);
            std::shared_ptr<CopyWork> copyWork = m_copyWorkPool.acquire();
            assert(req.u.write.count <= copyWork->hostMemory().size());

            GLOOP_CUDA_SAFE_CALL(hipMemcpyAsync(copyWork->hostMemory().hostPointer(), req.u.write.buffer, req.u.write.count, hipMemcpyDeviceToHost, copyWork->stream()));
            GLOOP_CUDA_SAFE_CALL(hipStreamSynchronize(copyWork->stream()));
            __sync_synchronize();

            ssize_t writtenCount = ::pwrite(req.u.write.fd, copyWork->hostMemory().hostPointer(), req.u.write.count, req.u.write.offset);

            m_copyWorkPool.release(copyWork);

            ipc->request()->u.writeResult.writtenCount = writtenCount;
            ipc->emit(Code::Complete);
        });
        break;
    }

    case Code::Read: {
        // FIXME: Significant naive implementaion.
        // We should integrate implementation with GPUfs's buffer cache.
        m_ioService.post([ipc, req, this]() {
            // GLOOP_DEBUG("Read ipc:(%p),fd:(%d),count:(%u),offset(%d),page:(%p)\n", (void*)ipc, req.u.read.fd, (unsigned)req.u.read.count, (int)req.u.read.offset, (void*)req.u.read.buffer);

            std::shared_ptr<CopyWork> copyWork = m_copyWorkPool.acquire();
            assert(req.u.read.count <= copyWork->hostMemory().size());
            ssize_t readCount = ::pread(req.u.read.fd, copyWork->hostMemory().hostPointer(), req.u.read.count, req.u.read.offset);
            __sync_synchronize();

            // FIXME: Should use multiple streams. And execute async.
            GLOOP_CUDA_SAFE_CALL(hipMemcpyAsync(req.u.read.buffer, copyWork->hostMemory().hostPointer(), readCount, hipMemcpyHostToDevice, copyWork->stream()));
            GLOOP_CUDA_SAFE_CALL(hipStreamSynchronize(copyWork->stream()));

            m_copyWorkPool.release(copyWork);

            ipc->request()->u.readResult.readCount = readCount;
            ipc->emit(Code::Complete);
        });
        break;
    }

    case Code::Fstat: {
        struct stat buf { };
        ::fstat(req.u.fstat.fd, &buf);
        // GLOOP_DEBUG("Fstat %d %u\n", req.u.fstat.fd, buf.st_size);
        ipc->request()->u.fstatResult.size = buf.st_size;
        ipc->emit(Code::Complete);
        break;
    }

    case Code::Close: {
        m_currentContext->table().close(req.u.close.fd);
        // GLOOP_DEBUG("Close %d\n", req.u.close.fd);
        ipc->request()->u.closeResult.error = 0;
        ipc->emit(Code::Complete);
        break;
    }

    case Code::Mmap: {
        // FIXME: Significant naive implementaion.
        // We should integrate implementation with GPUfs's buffer cache.
        m_ioService.post([ipc, req, this]() {
            // void* host = ::mmap(req.u.mmap.address, req.u.mmap.size, req.u.mmap.prot, req.u.mmap.flags, req.u.mmap.fd, req.u.mmap.offset);
            // void* host = ::mmap(req.u.mmap.address, req.u.mmap.size, req.u.mmap.prot, req.u.mmap.flags, MAP_ANONYMOUS, req.u.mmap.offset);
            void* host = ::mmap(req.u.mmap.address, req.u.mmap.size, req.u.mmap.prot, req.u.mmap.flags, req.u.mmap.fd, req.u.mmap.offset);
            GLOOP_DEBUG("mmap:address(%p),size:(%u),prot:(%d),flags:(%d),fd:(%d),offset:(%d),res:(%p)\n", req.u.mmap.address, req.u.mmap.size, req.u.mmap.prot, req.u.mmap.flags, req.u.mmap.fd, req.u.mmap.offset, host);
            void* device = nullptr;
            // volatile uint32_t value = *((volatile uint32_t*)host);
            // *((volatile uint32_t*)host) = value;
            GLOOP_CUDA_SAFE_CALL(hipHostRegister(host, req.u.mmap.size, hipHostRegisterMapped));
            GLOOP_CUDA_SAFE_CALL(hipHostGetDevicePointer(&device, host, 0));
            {
                std::lock_guard<HostContext::Mutex> guard(m_currentContext->mutex());
                m_currentContext->table().registerMapping(host, device);
                ipc->request()->u.mmapResult.address = device;
                ipc->emit(Code::ExitRequired);
                m_currentContext->addExitRequired(ipc);
            }
        });
        break;
    }

    case Code::Munmap: {
        // FIXME: Significant naive implementaion.
        // We should integrate implementation with GPUfs's buffer cache.
        m_ioService.post([ipc, req, this]() {
            GLOOP_DEBUG("munmap:address(%p),size:(%u)\n", req.u.munmap.address, req.u.munmap.size);
            // FIXME: We should schedule this inside this process.
            // GLOOP_CUDA_SAFE_CALL(hipHostUnregister((void*)req.u.munmap.address));
            GLOOP_DEBUG("Done\n");
            {
                std::lock_guard<HostContext::Mutex> guard(m_currentContext->mutex());
                void* host = m_currentContext->table().unregisterMapping((void*)req.u.munmap.address);
                int error = ::munmap(host, req.u.munmap.size);
                ipc->request()->u.munmapResult.error = error;
                ipc->emit(Code::ExitRequired);
                m_currentContext->addExitRequired(ipc);
            }
        });
        break;
    }

    case Code::Msync: {
        // FIXME: Significant naive implementaion.
        // We should integrate implementation with GPUfs's buffer cache.
        m_ioService.post([ipc, req, this]() {
            {
                std::lock_guard<HostContext::Mutex> guard(m_currentContext->mutex());
                void* host = m_currentContext->table().lookupHostByDevice((void*)req.u.msync.address);
                int error = ::msync(host, req.u.msync.size, req.u.msync.flags);
                ipc->request()->u.msyncResult.error = error;
                ipc->emit(Code::Complete);
            }
        });
        break;
    }

    }
    return false;
}

}  // namespace gloop
