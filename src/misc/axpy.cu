/*
  Copyright (C) 2015 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(expr) expr

template<typename Callback>
__global__ void axpy(float a, float* x, float* y, Callback callback) {
    callback(a, x, y);
    // y[threadIdx.x] = a * x[threadIdx.x];
}

int main(int argc, char* argv[]) {
    const int kDataLen = 4;

    float a = 2.0f;
    float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
    float host_y[kDataLen];

    // Copy input data to device.
    float* device_x;
    float* device_y;
    checkCudaErrors(hipMalloc(&device_x, kDataLen * sizeof(float)));
    checkCudaErrors(hipMalloc(&device_y, kDataLen * sizeof(float)));
    checkCudaErrors(hipMemcpy(device_x, host_x, kDataLen * sizeof(float), hipMemcpyHostToDevice));

    // Launch the kernel.
    // axpy<<<1, kDataLen>>>(a, device_x, device_y);
    axpy<<<1, kDataLen>>>(a, device_x, device_y, [=] (float a, float* x, float* y) {
        y[threadIdx.x] = a * x[threadIdx.x];
    });

    // Copy output data to host.
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(host_y, device_y, kDataLen * sizeof(float), hipMemcpyDeviceToHost));

    // Print the results.
    for (int i = 0; i < kDataLen; ++i) {
        std::cout << "y[" << i << "] = " << host_y[i] << "\n";
    }

    checkCudaErrors(hipDeviceReset());
    return 0;
}
