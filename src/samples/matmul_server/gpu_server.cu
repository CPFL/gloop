#include "hip/hip_runtime.h"
/*
  Copyright (C) 2016 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <gloop/gloop.h>
#include <gloop/benchmark.h>
#include "microbench_util.h"

#define THREADS_PER_TB 256
#define BLOCKS 1
#define BUF_SIZE 65536
#define NR_MSG   60000
#define MSG_SIZE BUF_SIZE

__device__ unsigned char g_message[512][MSG_SIZE];

__device__ void accept(gloop::DeviceLoop* loop, gloop::net::Server* server);

__device__ void close(gloop::DeviceLoop* loop, gloop::net::Server* server, gloop::net::Socket* socket)
{
    gloop::net::tcp::close(loop, socket, [=](gloop::DeviceLoop* loop, int error) {
        accept(loop, server);
    });
}

__device__ void perform(gloop::DeviceLoop* loop, gloop::net::Server* server, gloop::net::Socket* socket)
{
    gloop::net::tcp::receive(loop, socket, BUF_SIZE, g_message[gloop::logicalBlockIdx.x], [=](gloop::DeviceLoop* loop, ssize_t receiveCount) {
        if (receiveCount == 0) {
            close(loop, server, socket);
            return;
        }
        gloop::net::tcp::send(loop, socket, receiveCount, g_message[gloop::logicalBlockIdx.x], [=](gloop::DeviceLoop* loop, ssize_t sentCount) {
            if (sentCount == 0) {
                close(loop, server, socket);
                return;
            }
            perform(loop, server, socket);
        });
    });
}

__device__ void accept(gloop::DeviceLoop* loop, gloop::net::Server* server)
{
    gloop::net::tcp::accept(loop, server, [=](gloop::DeviceLoop* loop, gloop::net::Socket* socket) {
        if (!socket) {
            return;
        }
        perform(loop, server, socket);
    });
}

__device__ gloop::net::Server* globalServer = nullptr;
__device__ volatile gpunet::INIT_LOCK initLock;
__device__ void gpuMain(gloop::DeviceLoop* loop, struct sockaddr_in* addr)
{
    BEGIN_SINGLE_THREAD
    {
        __shared__ int toInit;
        toInit = initLock.try_wait();
        if (toInit == 1) {
            gloop::net::tcp::bind(loop, addr, [=](gloop::DeviceLoop* loop, gloop::net::Server* server) {
                assert(server);
                BEGIN_SINGLE_THREAD
                {
                    globalServer = server;
                    __threadfence();
                    initLock.signal();
                }
                END_SINGLE_THREAD
                accept(loop, globalServer);
            });
            return;
        }
    }
    END_SINGLE_THREAD
    accept(loop, globalServer);
}

int main(int argc, char** argv)
{
    dim3 blocks(BLOCKS);
    std::unique_ptr<gloop::HostLoop> hostLoop = gloop::HostLoop::create(0);
    std::unique_ptr<gloop::HostContext> hostContext = gloop::HostContext::create(*hostLoop, blocks);

    struct sockaddr* addr;
    struct sockaddr* dev_addr;
    {
        if (argc > 2) {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize, (2 << 20) * 256));
            gpunet_client_init(&addr, &dev_addr, argv[1], argv[2]);
            printf("address:(%x),port:(%u)\n", ((struct sockaddr_in*)addr)->sin_addr.s_addr, ((struct sockaddr_in*)addr)->sin_port);
        } else {
            gpunet_usage_client(argc, argv);
            exit(1);
        }
    }

    gloop::Benchmark benchmark;
    benchmark.begin();
    {
        hostLoop->launch(*hostContext, THREADS_PER_TB, [=] GLOOP_DEVICE_LAMBDA (gloop::DeviceLoop* loop, struct sockaddr* address) {
            gpuMain(loop, (struct sockaddr_in*)address);
        }, dev_addr);
    }
    benchmark.end();
    printf("[%d] ", 0);
    benchmark.report();

    return 0;
}
