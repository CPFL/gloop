#include "hip/hip_runtime.h"
/*
  Copyright (C) 2016 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <gloop/gloop.h>
#include <gloop/benchmark.h>
#include "microbench_util.h"

#define THREADS_PER_TB 256

__device__ void gpuMain(gloop::DeviceLoop* loop, struct sockaddr_in *addr) {
#if 0
	__shared__ int sock;
	int ret;
	sock = gconnect_in(addr);
	if (sock < 0) {
		BEGIN_SINGLE_THREAD_PART {
			gprintf4_single("ERROR: gconnect_in sock: %d", sock, 0, 0, 0);
		} END_SINGLE_THREAD_PART;
		return;
	}


	if (ret = gbench_send_recv_bw<BUF_SIZE, NR_MSG>(sock)) {
		printf("gbench_send_recv_bw ret: %d\n", ret);
		goto out;
	}

out:
	BEGIN_SINGLE_THREAD_PART {
		single_thread_gclose(sock);
	} END_SINGLE_THREAD_PART;
#endif
}

int main(int argc, char** argv)
{
    dim3 blocks(1);
    std::unique_ptr<gloop::HostLoop> hostLoop = gloop::HostLoop::create(0);
    std::unique_ptr<gloop::HostContext> hostContext = gloop::HostContext::create(*hostLoop, blocks);

    struct sockaddr* addr;
    struct sockaddr* dev_addr;
    {
        if (argc > 2) {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize, (2 << 20) * 256));
            gpunet_client_init(&addr, &dev_addr, argv[1], argv[2]);
        } else {
            gpunet_usage_client(argc, argv);
            exit(1);
        }
    }

    gloop::Benchmark benchmark;
    benchmark.begin();
    {
        hostLoop->launch(*hostContext, THREADS_PER_TB, [=] GLOOP_DEVICE_LAMBDA (gloop::DeviceLoop* loop, thrust::tuple<struct sockaddr*> tuple) {
            struct sockaddr* address;
            thrust::tie(address) = tuple;
            gpuMain(loop, (struct sockaddr_in*)address);
        }, dev_addr);
    }
    benchmark.end();
    printf("[%d] ", 0);
    benchmark.report();

    return 0;
}
