#include "hip/hip_runtime.h"
/*
  Copyright (C) 2016 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <gloop/gloop.h>
#include <gloop/benchmark.h>
#include <gloop/device_memory.cuh>


__device__ void perform_read(gloop::DeviceLoop* loop, uchar* scratch, int fd, size_t me, size_t filesize)
{
    if (me < filesize) {
        size_t toRead = min((size_t)GLOOP_SHARED_PAGE_SIZE, (size_t)(filesize - me));
        gloop::fs::read(loop, fd, me, toRead, scratch, [=](gloop::DeviceLoop* loop, int read) {
            if (toRead != read) {
                assert(NULL);
            }

            perform_read(loop, scratch, fd, me + GLOOP_SHARED_PAGE_SIZE * gloop::logicalGridDim.x, filesize);
        });
        return;
    }

    gloop::fs::close(loop, fd, [=](gloop::DeviceLoop* loop, int err) {
    });
}

__device__ void entry(gloop::DeviceLoop* loop, char* filename)
{
    __shared__ uchar* scratch;

    BEGIN_SINGLE_THREAD
    {
        scratch=(uchar*)malloc(GLOOP_SHARED_PAGE_SIZE);
        GLOOP_ASSERT(scratch!=NULL);
    }
    END_SINGLE_THREAD

    gloop::fs::open(loop, filename, O_RDONLY, [=](gloop::DeviceLoop* loop, int fd) {
        gloop::fs::fstat(loop, fd, [=](gloop::DeviceLoop* loop, int filesize) {
            size_t me = gloop::logicalBlockIdx.x * GLOOP_SHARED_PAGE_SIZE;
            perform_read(loop, scratch, fd, me, filesize);
        });
    });
}

int main(int argc, char** argv) {

    if(argc<5) {
        fprintf(stderr,"<kernel_iterations> <blocks> <threads> file\n");
        return -1;
    }
    int trials = atoi(argv[1]);
    int nblocks = atoi(argv[2]);
    int nthreads = atoi(argv[3]);

    fprintf(stderr," iterations: %d blocks %d threads %d\n",trials, nblocks, nthreads);

    {
        dim3 blocks(nblocks);
        std::unique_ptr<gloop::HostLoop> hostLoop = gloop::HostLoop::create(0);
        std::unique_ptr<gloop::HostContext> hostContext = gloop::HostContext::create(*hostLoop, blocks);
        const std::string filename(argv[4]);

        std::shared_ptr<gloop::DeviceMemory> memory = gloop::DeviceMemory::create(filename.size() + 1);
        CUDA_SAFE_CALL(hipMemcpy(memory->devicePointer(), filename.c_str(), filename.size() + 1,hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize, (1ULL << 20) * 896));

        gloop::Benchmark bench;
        bench.begin();
        hostLoop->launch(*hostContext, nthreads, [=] GLOOP_DEVICE_LAMBDA (gloop::DeviceLoop* loop, char* filename) {
            entry(loop, filename);
        }, reinterpret_cast<char*>(memory->devicePointer()));
        bench.end();
        bench.report();
    }

    return 0;
}
