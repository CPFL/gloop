#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Includes
////////////////////////////////////////////////////////////////////////////////
#include "mergesort.cuh"
#include "mergesort_inlines.cuh"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <gloop/sync_read_write.h>
////////////////////////////////////////////////////////////////////////////////
// Defines
////////////////////////////////////////////////////////////////////////////////
#define BLOCKSIZE 256
#define ROW_LENGTH BLOCKSIZE * 4
#define ROWS 4096

struct MergeSortContext {
    float4 a;
    float4 b;
    int aidx;
    int bidx;
    int disabled;
};

static __device__ void destroyContext(Context ctx, MergeSortContext* context)
{
    BEGIN_SINGLE_THREAD
    {
        delete [] context;
        ctx.device->contexts[blockIdx.x] = nullptr;
    }
    END_SINGLE_THREAD
}

static __device__ void mergeSortPassKernel(Context ctx, MergeSortContext* context, float4* result, int nrElems, int threadsPerDiv, int outidx, int Astart, int Bstart, int tid, int division, int aidx, int bidx, float4 a, float4 b, int disabled)
{
    float4* resStart = result + Astart;
    while (true) //aidx < nrElems/2)// || (bidx < nrElems/2  && (Bstart + bidx < constEndAddr[division])))
    {
        float4 nextA;
        float4 nextB;
        bool elemsLeftInA = false;
        bool elemsLeftInB = false;
        if (!disabled) {
            /**
             * For some reason, it's faster to do the texture fetches here than
             * after the merge
             */
            nextA = tex1Dfetch(tex, Astart + aidx + 1);
            nextB = tex1Dfetch(tex, Bstart + bidx + 1);

            float4 na = getLowest(a, b);
            float4 nb = getHighest(a, b);
            a = sortElem(na);
            b = sortElem(nb);
            // Now, a contains the lowest four elements, sorted
            resStart[outidx] = a;

            elemsLeftInA = (aidx + 1 < nrElems / 2); // Astart + aidx + 1 is allways less than division border
            elemsLeftInB = (bidx + 1 < nrElems / 2) && (Bstart + bidx + 1 < constStartAddr[division + 1]);

            if (elemsLeftInA) {
                if (elemsLeftInB) {
                    if (nextA.x < nextB.x) {
                        aidx += 1;
                        a = nextA;
                    } else {
                        bidx += 1;
                        a = nextB;
                    }
                } else {
                    aidx += 1;
                    a = nextA;
                }
            } else {
                if (elemsLeftInB) {
                    bidx += 1;
                    a = nextB;
                }
            }
        }
        outidx++;

        if (__syncthreads_and(!elemsLeftInA && !elemsLeftInB))
            break;

        context[threadIdx.x].aidx = aidx;
        context[threadIdx.x].bidx = bidx;
        context[threadIdx.x].a = a;
        context[threadIdx.x].b = b;
        context[threadIdx.x].disabled = disabled;
        gloop::syncWrite<unsigned int>(ctx.continuing, 1);
        return;
    }

    if (!disabled) {
        resStart[outidx++] = b;
    }
    destroyContext(ctx, context);
}

static __global__ void mergeSortPassSecondKernel(Context ctx, float4* result, int nrElems, int threadsPerDiv, int outidx)
{
    __shared__ MergeSortContext* context;
    BEGIN_SINGLE_THREAD
    {
        context = ctx.device->contexts[blockIdx.x];
    }
    END_SINGLE_THREAD

    if (context == nullptr)
        return;

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int division = tid / threadsPerDiv;
    int int_tid = tid - division * threadsPerDiv;
    int disabled = context[threadIdx.x].disabled;
    int Astart = 0;
    int Bstart = 0;
    if (!disabled) {
        Astart = constStartAddr[division] + int_tid * nrElems;
        Bstart = Astart + nrElems / 2;
    }
    mergeSortPassKernel(ctx, context, result, nrElems, threadsPerDiv, outidx, Astart, Bstart, tid, division, context[threadIdx.x].aidx, context[threadIdx.x].bidx, context[threadIdx.x].a, context[threadIdx.x].b, disabled);
}

static __global__ void mergeSortPassInitialKernel(Context ctx, float4* result, int nrElems, int threadsPerDiv)
{
    __shared__ MergeSortContext* context;
#if 1
    BEGIN_SINGLE_THREAD
    {
        context = ctx.device->contexts[blockIdx.x] = new MergeSortContext[blockDim.x];
        GPU_ASSERT(context);
    }
    END_SINGLE_THREAD
#endif
    if (context == nullptr)
        return;

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    // The division to work on
    int division = tid / threadsPerDiv;
    int disabled = 0;
    int Astart = 0;
    int Bstart = 0;
    float4 a;
    float4 b;

    if (division >= DIVISIONS) {
        disabled = 1;
    } else {
        // The block within the division
        int int_tid = tid - division * threadsPerDiv;
        Astart = constStartAddr[division] + int_tid * nrElems;

        Bstart = Astart + nrElems / 2;
        float4* resStart = &(result[Astart]);

        if (Astart >= constStartAddr[division + 1]) {
            disabled = 1;
        } else {
            if (Bstart >= constStartAddr[division + 1]) {
                for (int i = 0; i < (constStartAddr[division + 1] - Astart); i++) {
                    resStart[i] = tex1Dfetch(tex, Astart + i);
                }
                disabled = 1;
            } else {
                a = tex1Dfetch(tex, Astart);
                b = tex1Dfetch(tex, Bstart);
            }
        }
    }

    mergeSortPassKernel(ctx, context, result, nrElems, threadsPerDiv, 0, Astart, Bstart, tid, division, 0, 0, a, b, disabled);
}

////////////////////////////////////////////////////////////////////////////////
// The mergesort algorithm
////////////////////////////////////////////////////////////////////////////////
float4* runMergeSort(Context* ctx, int listsize, int divisions,
    float4* d_origList, float4* d_resultList,
    int* sizes, int* nullElements,
    unsigned int* origOffsets)
{
    int* startaddr = (int*)malloc((divisions + 1) * sizeof(int));
    int largestSize = -1;
    startaddr[0] = 0;
    for (int i = 1; i <= divisions; i++) {
        startaddr[i] = startaddr[i - 1] + sizes[i - 1];
        if (sizes[i - 1] > largestSize)
            largestSize = sizes[i - 1];
    }
    largestSize *= 4;

    // Setup texture
    hipChannelFormatDesc channelDesc;
    {
        channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
        tex.addressMode[0] = hipAddressModeWrap;
        tex.addressMode[1] = hipAddressModeWrap;
        tex.filterMode = hipFilterModePoint;
        tex.normalized = false;
    }

////////////////////////////////////////////////////////////////////////////
// First sort all float4 elements internally
////////////////////////////////////////////////////////////////////////////
#ifdef MERGE_WG_SIZE_0
    const int THREADS = MERGE_WG_SIZE_0;
#else
    const int THREADS = 256;
#endif
    dim3 threads(THREADS, 1);
    int blocks = ((listsize / 4) % THREADS == 0) ? (listsize / 4) / THREADS : (listsize / 4) / THREADS + 1;
    dim3 grid(blocks, 1);
    {
        hipBindTexture(0, tex, d_origList, channelDesc, listsize * sizeof(float));
    }
    mergeSortFirst(ctx, grid, threads, d_resultList, listsize);

    ////////////////////////////////////////////////////////////////////////////
    // Then, go level by level
    ////////////////////////////////////////////////////////////////////////////
    {
        hipMemcpyToSymbol(HIP_SYMBOL(constStartAddr), startaddr, (divisions + 1) * sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(finalStartAddr), origOffsets, (divisions + 1) * sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(nullElems), nullElements, (divisions) * sizeof(int));
    }
    int nrElems = 2;
    while (true) {
        int floatsperthread = (nrElems * 4);
        int threadsPerDiv = (int)ceil(largestSize / (float)floatsperthread);
        int threadsNeeded = threadsPerDiv * divisions;
#ifdef MERGE_WG_SIZE_1
        threads.x = MERGE_WG_SIZE_1;
#else
        threads.x = 208;
#endif
        grid.x = ((threadsNeeded % threads.x) == 0) ? threadsNeeded / threads.x : (threadsNeeded / threads.x) + 1;
        if (grid.x < 8) {
            grid.x = 8;
            threads.x = ((threadsNeeded % grid.x) == 0) ? threadsNeeded / grid.x : (threadsNeeded / grid.x) + 1;
        }
        // Swap orig/result list
        float4* tempList = d_origList;
        d_origList = d_resultList;
        d_resultList = tempList;
        {
            hipBindTexture(0, tex, d_origList, channelDesc, listsize * sizeof(float));
        }

        gloop::syncWrite<unsigned int>(ctx->continuing, 0);
        mergeSortPassInitialKernel<<<grid, threads>>>(*ctx, d_resultList, nrElems, threadsPerDiv);
        hipDeviceSynchronize();

#if 1
        for (int i = 0; gloop::readNoCache<unsigned int>(ctx->continuing); ++i) {
            gloop::syncWrite<unsigned int>(ctx->continuing, 0);
            mergeSortPassSecondKernel<<<grid, threads>>>(*ctx, d_resultList, nrElems, threadsPerDiv, i);
            hipDeviceSynchronize();
        }
#endif

        nrElems *= 2;
        floatsperthread = (nrElems * 4);
        if (threadsPerDiv == 1)
            break;
    }
////////////////////////////////////////////////////////////////////////////
// Now, get rid of the NULL elements
////////////////////////////////////////////////////////////////////////////
#ifdef MERGE_WG_SIZE_0
    threads.x = MERGE_WG_SIZE_0;
#else
    threads.x = 256;
#endif
    grid.x = ((largestSize % threads.x) == 0) ? largestSize / threads.x : (largestSize / threads.x) + 1;
    grid.y = divisions;
    mergepack(ctx, grid, threads, (float*)d_resultList, (float*)d_origList);

    free(startaddr);
    return d_origList;
}
