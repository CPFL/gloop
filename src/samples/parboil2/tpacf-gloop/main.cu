/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include <gloop/benchmark.h>
#include <gloop/gloop.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "args.h"

#include "model.h"
#include "tpacf_kernel.cu"

#define CUDA_ERRCK { hipError_t err; \
    if ((err = hipGetLastError()) != hipSuccess) { \
        printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
        return -1; }}

extern unsigned int NUM_SETS;
extern unsigned int NUM_ELEMENTS;

int main( int argc, char** argv)
{
    struct pb_TimerSet timers;
    struct pb_Parameters *params;

    {
        pb_InitializeTimerSet( &timers );
        params = pb_ReadParameters( &argc, argv );

        options args;
        parse_args(argc, argv, &args);

        pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

        NUM_ELEMENTS = args.npoints;
        NUM_SETS = args.random_count;
        int num_elements = NUM_ELEMENTS;

        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(NUM_SETS*2 + 1);
        std::unique_ptr<gloop::HostLoop> hostLoop = gloop::HostLoop::create(0);
        std::unique_ptr<gloop::HostContext> hostContext = gloop::HostContext::create(*hostLoop, dimGrid, dimGrid);

        printf("Min distance: %f arcmin\n", min_arcmin);
        printf("Max distance: %f arcmin\n", max_arcmin);
        printf("Bins per dec: %i\n", bins_per_dec);
        printf("Total bins  : %i\n", NUM_BINS);

        //read in files
        unsigned mem_size = (1+NUM_SETS)*num_elements*sizeof(struct cartesian);
        unsigned f_mem_size = (1+NUM_SETS)*num_elements*sizeof(REAL);

        // container for all the points read from files
        struct cartesian *h_all_data;
        h_all_data = (struct cartesian*) malloc(mem_size);
        // Until I can get libs fixed

        // iterator for data files
        struct cartesian *working = h_all_data;

        // go through and read all data and random points into h_all_data
        pb_SwitchToTimer( &timers, pb_TimerID_IO );
        readdatafile(params->inpFiles[0], working, num_elements);
        pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

        working += num_elements;
        for(int i = 0; i < (NUM_SETS); i++)
        {
            pb_SwitchToTimer( &timers, pb_TimerID_IO );
            readdatafile(params->inpFiles[i+1], working, num_elements);
            pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

            working += num_elements;
        }

        // split into x, y, and z arrays
        REAL * h_x_data = (REAL*) malloc (3*f_mem_size);
        REAL * h_y_data = h_x_data + NUM_ELEMENTS*(NUM_SETS+1);
        REAL * h_z_data = h_y_data + NUM_ELEMENTS*(NUM_SETS+1);
        for(int i = 0; i < (NUM_SETS+1); ++i)
        {
            for(int j = 0; j < NUM_ELEMENTS; ++j)
            {
                h_x_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].x;
                h_y_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].y;
                h_z_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].z;
            }
        }

        // from on use x, y, and z arrays, free h_all_data
        free(h_all_data);

        // allocate cuda memory to hold all points
        hist_t* new_hists;
        hist_t* d_hists;
        REAL* d_x_data;
        REAL * d_y_data;
        REAL * d_z_data;
        {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());

            pb_SwitchToTimer( &timers, pb_TimerID_COPY );
            hipMalloc((void**) & d_x_data, 3*f_mem_size);
            CUDA_ERRCK
            d_y_data = d_x_data + NUM_ELEMENTS*(NUM_SETS+1);
            d_z_data = d_y_data + NUM_ELEMENTS*(NUM_SETS+1);

            // allocate cuda memory to hold final histograms
            // (1 for dd, and NUM_SETS for dr and rr apiece)
            hipMalloc((void**) & d_hists, NUM_BINS*(NUM_SETS*2+1)*sizeof(hist_t) );
            CUDA_ERRCK
            pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

            // allocate system memory for final histograms
            new_hists = (hist_t *) malloc(NUM_BINS*(NUM_SETS*2+1)* sizeof(hist_t));

            // Initialize the boundary constants for bin search
            initBinB( &timers );
            CUDA_ERRCK

            // **===------------------ Kick off TPACF on CUDA------------------===**
            pb_SwitchToTimer( &timers, pb_TimerID_COPY );
            hipMemcpy(d_x_data, h_x_data, 3*f_mem_size, hipMemcpyHostToDevice);
            CUDA_ERRCK
            pb_SwitchToTimer( &timers, pb_TimerID_KERNEL );
        }

        {
            // FIXME.
            // gloop::Benchmark benchmark;
            // hipDeviceSynchronize();
            // benchmark.begin();
            hostLoop->launch(*hostContext, dimBlock, [=] GLOOP_DEVICE_LAMBDA (gloop::DeviceLoop* loop, hist_t* histograms, REAL* all_x_data, REAL* all_y_data, REAL* all_z_data, unsigned int NUM_SETS, unsigned int NUM_ELEMENTS) {
                gen_hists(loop, histograms, all_x_data, all_y_data, all_z_data, NUM_SETS, NUM_ELEMENTS);
            }, d_hists, d_x_data, d_y_data, d_z_data, NUM_SETS, NUM_ELEMENTS);
            // FIXME.
            // hipDeviceSynchronize();
            // benchmark.end();
            // benchmark.report();
        }

        {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            pb_SwitchToTimer( &timers, pb_TimerID_COPY );
            hipMemcpy(new_hists, d_hists, NUM_BINS*(NUM_SETS*2+1)* sizeof(hist_t), hipMemcpyDeviceToHost);
            CUDA_ERRCK
            pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
        }
        // **===-----------------------------------------------------------===**

        // references into output histograms
        hist_t *dd_hist = new_hists;
        hist_t *rr_hist = dd_hist + NUM_BINS;
        hist_t *dr_hist = rr_hist + NUM_BINS*NUM_SETS;

        // add up values within dr and rr
        int rr[NUM_BINS];
        for(int i=0; i<NUM_BINS; i++)
        {
            rr[i] = 0;
        }
        for(int i=0; i<NUM_SETS; i++)
        {
            for(int j=0; j<NUM_BINS; j++)
            {
                rr[j] += rr_hist[i*NUM_BINS + j];
            }
        }
        int dr[NUM_BINS];
        for(int i=0; i<NUM_BINS; i++)
        {
            dr[i] = 0;
        }
        for(int i=0; i<NUM_SETS; i++)
        {
            for(int j=0; j<NUM_BINS; j++)
            {
                dr[j] += dr_hist[i*NUM_BINS + j];
            }
        }

        //int dd_t = 0;
        //int dr_t = 0;
        //int rr_t = 0;
        FILE *outfile;
        if ((outfile = fopen(params->outFile, "w")) == NULL)
        {
            fprintf(stderr, "Unable to open output file %s for writing, "
                    "assuming stdout\n", params->outFile);
            outfile = stdout;
        }

        {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            pb_SwitchToTimer( &timers, pb_TimerID_IO );
            // print out final histograms + omega (while calculating omega)
            for(int i=0; i<NUM_BINS; i++)
            {
                //REAL w = (100.0 * dd_hist[i] - dr[i]) / rr[i] + 1.0;
                //fprintf(outfile, "%f\n", w);
                fprintf(outfile, "%d\n%d\n%d\n", dd_hist[i], dr[i], rr[i]);
                //      dd_t += dd_hist[i];
                //      dr_t += dr[i];
                //      rr_t += rr[i];
            }
            pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
        }

        if(outfile != stdout)
            fclose(outfile);

        // cleanup memory
        free(new_hists);
        free( h_x_data);

        {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            pb_SwitchToTimer( &timers, pb_TimerID_COPY );
            hipFree( d_hists );
            hipFree( d_x_data );
            pb_SwitchToTimer(&timers, pb_TimerID_NONE);
            pb_PrintTimerSet(&timers);
            pb_FreeParameters(params);
        }
    }
}

