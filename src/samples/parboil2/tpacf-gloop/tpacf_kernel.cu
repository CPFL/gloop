#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

#include <assert.h>
#include "model.h"
#include <math.h>

#define WARP_SIZE 32
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define BLOCK_SIZE 256
#define NUM_WARPS (BLOCK_SIZE/WARP_SIZE)
#define HISTS_PER_WARP 16
#define NUM_HISTOGRAMS  (NUM_WARPS*HISTS_PER_WARP)
#define THREADS_PER_HIST (WARP_SIZE/HISTS_PER_WARP)

REAL** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

__constant__ REAL dev_binb[NUM_BINS+1];

unsigned int NUM_SETS;
unsigned int NUM_ELEMENTS;

// create the bin boundaries
void initBinB( struct pb_TimerSet *timers )
{
    REAL *binb = (REAL*)malloc((NUM_BINS+1)*sizeof(REAL));
    for (int k = 0; k < NUM_BINS+1; k++) {
        binb[k] = cos(pow(10.0, (log10(min_arcmin) + k*1.0/bins_per_dec)) / 60.0*D2R);
    }
    pb_SwitchToTimer( timers, pb_TimerID_COPY );
    hipMemcpyToSymbol(HIP_SYMBOL(dev_binb), binb, (NUM_BINS+1)*sizeof(REAL));
    pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
    free(binb);
}

template<typename Callback>
__device__ void iterateOverAllRandomPoints(gloop::DeviceLoop* loop, struct cartesian* data, bool do_self, REAL* random_x, REAL* random_y, REAL* random_z, unsigned int (*warp_hists)[NUM_HISTOGRAMS], unsigned int NUM_ELEMENTS, unsigned int i, unsigned int j, Callback callback)
{
    // Iterate over all random points
    if (j < NUM_ELEMENTS) {
        // load current random point values
        REAL random_x_s;
        REAL random_y_s;
        REAL random_z_s;

        if(threadIdx.x + j < NUM_ELEMENTS) {
            random_x_s = random_x[threadIdx.x + j];
            random_y_s = random_y[threadIdx.x + j];
            random_z_s = random_z[threadIdx.x + j];
        }

        // Iterate for all elements of current set of data points
        // (BLOCK_SIZE iterations per thread)
        // Each thread calcs against 1 random point within cur set of random
        // (so BLOCK_SIZE threads covers all random points within cur set)
        for(unsigned int k = 0; (k < BLOCK_SIZE) && (k+i < NUM_ELEMENTS); k += 1) {
            // do actual calculations on the values:
            REAL distance =
                data[k].x * random_x_s +
                data[k].y * random_y_s +
                data[k].z * random_z_s;

            unsigned int bin_index;

            // run binary search to find bin_index
            unsigned int min = 0;
            unsigned int max = NUM_BINS;
            {
                unsigned int k2;

                while (max > min+1) {
                    k2 = (min + max) / 2;
                    if (distance >= dev_binb[k2])
                        max = k2;
                    else
                        min = k2;
                }
                bin_index = max - 1;
            }

            unsigned int warpnum = threadIdx.x / (WARP_SIZE/HISTS_PER_WARP);
            if((distance < dev_binb[min]) && (distance >= dev_binb[max]) &&
                    (!do_self || (threadIdx.x + j > i + k)) && (threadIdx.x + j < NUM_ELEMENTS)) {
                atomicAdd(&warp_hists[bin_index][warpnum], 1U);
            }
        }

        gloop::loop::postTask(loop, [=] (gloop::DeviceLoop* loop) {
            iterateOverAllRandomPoints(loop, data, do_self, random_x, random_y, random_z, warp_hists, NUM_ELEMENTS, i, j + BLOCK_SIZE, callback);
        });
        return;
    }

    callback(loop);
}

template<typename Callback>
__device__ void iterateOverAllDataPoints(gloop::DeviceLoop* loop, struct cartesian* data, REAL* data_x, REAL* data_y, REAL* data_z, bool do_self, REAL* random_x, REAL* random_y, REAL* random_z, unsigned int (*warp_hists)[NUM_HISTOGRAMS], unsigned int NUM_ELEMENTS, unsigned int i, Callback callback)
{
    // Iterate over all data points
    if (i < NUM_ELEMENTS) {
        // load current set of data into shared memory
        // (total of BLOCK_SIZE points loaded)
        if(threadIdx.x + i < NUM_ELEMENTS) {
            // reading outside of bounds is a-okay
            data[threadIdx.x] = (struct cartesian) {data_x[threadIdx.x + i], data_y[threadIdx.x + i], data_z[threadIdx.x + i]};
        }

        __syncthreads();

        iterateOverAllRandomPoints(loop, data, do_self, random_x, random_y, random_z, warp_hists, NUM_ELEMENTS, i, (do_self ? i+1 : 0), [=] (gloop::DeviceLoop* loop) {
            iterateOverAllDataPoints(loop, data, data_x, data_y, data_z, do_self, random_x, random_y, random_z, warp_hists, NUM_ELEMENTS, i + BLOCK_SIZE, callback);
        });
        return;
    }

    callback(loop);
}


__device__ void gen_hists(gloop::DeviceLoop* loop, hist_t* histograms, REAL* all_x_data, REAL* all_y_data, REAL* all_z_data, int NUM_SETS, int NUM_ELEMENTS)
{
    unsigned int bx = gloop::logicalBlockIdx.x;
    unsigned int tid = threadIdx.x;
    bool do_self = (bx < (NUM_SETS + 1));

    REAL* data_x;
    REAL* data_y;
    REAL* data_z;
    REAL* random_x;
    REAL* random_y;
    REAL* random_z;

    __shared__ struct cartesian* data;
    __shared__ unsigned int (*warp_hists)[NUM_HISTOGRAMS];
    BEGIN_SINGLE_THREAD
    {
        data = new struct cartesian[BLOCK_SIZE];
        warp_hists = new unsigned int[NUM_BINS][NUM_HISTOGRAMS]; // 640B <1k
    }
    END_SINGLE_THREAD

    for(unsigned int w = 0; w < NUM_BINS*NUM_HISTOGRAMS; w += BLOCK_SIZE) {
        if(w+tid < NUM_BINS*NUM_HISTOGRAMS) {
            warp_hists[(w+tid)/NUM_HISTOGRAMS][(w+tid)%NUM_HISTOGRAMS] = 0;
        }
    }

    // Get stuff into shared memory to kick off the loop.
    if(!do_self) {
        data_x = all_x_data;
        data_y = all_y_data;
        data_z = all_z_data;
        random_x = all_x_data + NUM_ELEMENTS * (bx - NUM_SETS);
        random_y = all_y_data + NUM_ELEMENTS * (bx - NUM_SETS);
        random_z = all_z_data + NUM_ELEMENTS * (bx - NUM_SETS);
    } else {
        random_x = all_x_data + NUM_ELEMENTS * (bx);
        random_y = all_y_data + NUM_ELEMENTS * (bx);
        random_z = all_z_data + NUM_ELEMENTS * (bx);

        data_x = random_x;
        data_y = random_y;
        data_z = random_z;
    }

    gloop::loop::postTask(loop, [=] (gloop::DeviceLoop* loop) {
        iterateOverAllDataPoints(loop, data, data_x, data_y, data_z, do_self, random_x, random_y, random_z, warp_hists, NUM_ELEMENTS, 0, [=] (gloop::DeviceLoop* loop) {
            gloop::loop::postTask(loop, [=] (gloop::DeviceLoop* loop) {
                // coalesce the histograms in a block
                unsigned int warp_index = tid & ( (NUM_HISTOGRAMS>>1) - 1);
                unsigned int bin_index = tid / (NUM_HISTOGRAMS>>1);
                for(unsigned int offset = NUM_HISTOGRAMS >> 1; offset > 0; offset >>= 1) {
                    for(unsigned int bin_base = 0; bin_base < NUM_BINS; bin_base += BLOCK_SIZE/ (NUM_HISTOGRAMS>>1)) {
                        __syncthreads();
                        if(warp_index < offset && bin_base+bin_index < NUM_BINS ) {
                            unsigned long sum =
                                warp_hists[bin_base + bin_index][warp_index] +
                                warp_hists[bin_base + bin_index][warp_index+offset];
                            warp_hists[bin_base + bin_index][warp_index] = sum;
                        }
                    }
                }

                __syncthreads();

                // Put the results back in the real histogram
                // warp_hists[x][0] holds sum of all locations of bin x
                hist_t* hist_base = histograms + NUM_BINS * bx;
                if(tid < NUM_BINS) {
                    hist_base[tid] = warp_hists[tid][0];
                }

                BEGIN_SINGLE_THREAD
                {
                    delete [] data;
                    delete [] warp_hists;
                }
                END_SINGLE_THREAD
            });
        });
    });
}

// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
