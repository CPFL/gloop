#include "hip/hip_runtime.h"
/*
  Copyright (C) 2016 Yusuke Suzuki <yusuke.suzuki@sslab.ics.keio.ac.jp>

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
  ARE DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
  THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <gloop/gloop.h>
#include <gloop/benchmark.h>
#include "microbench_util.h"

#define THREADS_PER_TB 256
#define BLOCKS 16
#define BUF_SIZE 65536
#define NR_MSG   60000
#define MSG_SIZE BUF_SIZE

class EchoServer {
public:
    __device__ EchoServer(gloop::net::Server* server)
        : m_server(server)
    {
    }

    __device__ void accept(gloop::DeviceLoop<>* loop)
    {
        gloop::net::tcp::accept(loop, m_server, [=](gloop::DeviceLoop<>* loop, gloop::net::Socket* socket) {
            if (!socket) {
                return;
            }
            this->handle(loop, socket);
        });
    }

    __device__ void close(gloop::DeviceLoop<>* loop, gloop::net::Socket* socket)
    {
        gloop::net::tcp::close(loop, socket, [=](gloop::DeviceLoop<>* loop, int error) {
            this->accept(loop);
        });
    }

    __device__ void handle(gloop::DeviceLoop<>* loop, gloop::net::Socket* socket)
    {
        gloop::net::tcp::receive(loop, socket, BUF_SIZE, (uint8_t*)m_message, 0, [=](gloop::DeviceLoop<>* loop, ssize_t receiveCount) {
            if (receiveCount == 0) {
                this->close(loop, socket);
                return;
            }
            gloop::net::tcp::send(loop, socket, receiveCount, (uint8_t*)m_message, [=](gloop::DeviceLoop<>* loop, ssize_t sentCount) {
                if (sentCount == 0) {
                    this->close(loop, socket);
                    return;
                }
                this->handle(loop, socket);
            });
        });
    }

private:
    unsigned char m_message[BUF_SIZE];
    gloop::net::Server* m_server;
};

__device__ gloop::net::Server* globalServer = nullptr;
__device__ volatile gpunet::INIT_LOCK initLock;
__device__ void gpuMain(gloop::DeviceLoop<>* loop, struct sockaddr_in* addr)
{
    __shared__ EchoServer* echoServer;
    __shared__ int toInit;
    BEGIN_SINGLE_THREAD
    {
        toInit = initLock.try_wait();
        if (toInit != 1)
            echoServer = new EchoServer(globalServer);
    }
    END_SINGLE_THREAD
    if (toInit == 1) {
        gloop::net::tcp::bind(loop, addr, [=](gloop::DeviceLoop<>* loop, gloop::net::Server* server) {
            assert(server);
            __shared__ EchoServer* echoServer;
            BEGIN_SINGLE_THREAD
            {
                globalServer = server;
                __threadfence();
                initLock.signal();
                echoServer = new EchoServer(globalServer);
            }
            END_SINGLE_THREAD
            echoServer->accept(loop);
        });
        return;
    }
    echoServer->accept(loop);
}

int main(int argc, char** argv)
{
    dim3 blocks(BLOCKS);
    std::unique_ptr<gloop::HostLoop> hostLoop = gloop::HostLoop::create(0);
    std::unique_ptr<gloop::HostContext> hostContext = gloop::HostContext::create(*hostLoop, blocks);

    struct sockaddr* addr;
    struct sockaddr* dev_addr;
    {
        if (argc > 2) {
            std::lock_guard<gloop::HostLoop::KernelLock> lock(hostLoop->kernelLock());
            CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize, (2 << 20) * 256));
            gpunet_client_init(&addr, &dev_addr, argv[1], argv[2]);
            printf("address:(%x),port:(%u)\n", ((struct sockaddr_in*)addr)->sin_addr.s_addr, ((struct sockaddr_in*)addr)->sin_port);
        } else {
            gpunet_usage_client(argc, argv);
            exit(1);
        }
    }

    gloop::Benchmark benchmark;
    benchmark.begin();
    {
        hostLoop->launch(*hostContext, blocks, THREADS_PER_TB, [=] GLOOP_DEVICE_LAMBDA (gloop::DeviceLoop<>* loop, struct sockaddr* address) {
            gpuMain(loop, (struct sockaddr_in*)address);
        }, dev_addr);
    }
    benchmark.end();
    printf("[%d] ", 0);
    benchmark.report();

    return 0;
}
